#include "hip/hip_runtime.h"
#include <c10/cuda/CUDAException.h>
#include <c10/cuda/CUDAStream.h>
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void flash_attn_forward_kernel(
    const float *__restrict__ q,
    const float *__restrict__ k,
    const float *__restrict__ v,
    float *__restrict__ out,
    int T_r,
    int T_c,
    float tau,
    float *l_hbm,
    float *m_hbm,
    int b,
    int h,
    int s,
    int d_head,
    int B_r,
    int B_c)
{
    int block_b = blockIdx.x;   // batch index
    int block_h = blockIdx.y;   // head index
    int thread_i = threadIdx.x; // thread index within the block

    extern __shared__ float shared_memory[]; // Dynamic shared memory
    float *K_shared = shared_memory;
    float *V_shared = &shared_memory[B_c * d_head];
    float *Q_shared = &shared_memory[(B_c + B_c) * d_head];

    for (int j = 0; j < T_r; j++)
    {
        int r_idx = j * B_r + thread_i;
        if (r_idx < s)
        {
            for (int c = 0; c < d_head; c++)
            {
                Q_shared[thread_i * d_head + c] = q[block_b * h * s * d_head + block_h * s * d_head + r_idx * d_head + c];
            }
        }
        __syncthreads();

        float l = 0.0f, m = 0.0f;
        for (int i = 0; i < T_c; i++)
        {
            int c_idx = i * B_c + thread_i;
            if (c_idx < s)
            {
                for (int c = 0; c < d_head; c++)
                {
                    K_shared[thread_i * d_head + c] = k[block_b * h * s * d_head + block_h * s * d_head + c_idx * d_head + c];
                    V_shared[thread_i * d_head + c] = v[block_b * h * s * d_head + block_h * s * d_head + c_idx * d_head + c];
                }
            }
            __syncthreads();

            if (r_idx < s)
            {
                for (int b_c = i * B_c; b_c < min((i + 1) * B_c, s); b_c++)
                {
                    int b_c_local = b_c - i * B_c;
                    float curr_l = l;
                    float curr_m = m;
                    float Sij = 0.0f;

                    for (int k_dim = 0; k_dim < d_head; k_dim++)
                    {
                        Sij += tau * (Q_shared[thread_i * d_head + k_dim] * K_shared[b_c_local * d_head + k_dim]);
                    }

                    float new_m = fmaxf(curr_m, Sij);
                    float exp_Sij = expf(Sij - new_m);
                    float exp_max = expf(curr_m - new_m);
                    float new_l = curr_l * exp_max + exp_Sij;

                    // Update output Oi += softmax * Vj
                    for (int v_dim = 0; v_dim < d_head; v_dim++)
                    {
                        out[block_b * h * s * d_head + block_h * s * d_head + r_idx * d_head + v_dim] =
                            out[block_b * h * s * d_head + block_h * s * d_head + r_idx * d_head + v_dim] * (curr_l * exp_max / new_l) + (exp_Sij / new_l) * V_shared[b_c_local * d_head + v_dim];
                    }

                    l = new_l;
                    m = new_m;
                }
            }
            __syncthreads();
        }

        if (r_idx < s)
        {
            l_hbm[block_b * h * s + block_h * s + r_idx] = l;
            m_hbm[block_b * h * s + block_h * s + r_idx] = m;
        }
    }
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> flash_attention_forwad(torch::Tensor q, torch::Tensor k, torch::Tensor v, float tau)
{
    int b = q.size(0);
    int h = q.size(1);
    int s = q.size(2);
    int d_head = q.size(3);

    int device = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    int maxSharedMemPerBlock;
    hipDeviceGetAttribute(&maxSharedMemPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, device);

    int B_r = maxSharedMemPerBlock / (3 * d_head * sizeof(float));
    int B_c = maxSharedMemPerBlock / (3 * d_head * sizeof(float));

    int T_r = (s + B_r - 1) / B_r;
    int T_c = (s + B_c - 1) / B_c;

    torch::Tensor out = torch::zeros_like(q);

    dim3 grid(b, h);
    dim3 block(B_r);
    int shared_mem_size = (B_c * d_head * sizeof(float)) * 3; 

    auto l_hbm = torch::zeros({b, h, s}, torch::CUDA(torch::kFloat32));
    auto m_hbm = torch::zeros({b, h, s}, torch::CUDA(torch::kFloat32));

    flash_attn_forward_kernel<<<grid, block, shared_mem_size>>>(
        q.data_ptr<float>(),
        k.data_ptr<float>(),
        v.data_ptr<float>(),
        out.data_ptr<float>(),
        T_r,
        T_c,
        tau,
        l_hbm.data_ptr<float>(),
        m_hbm.data_ptr<float>(),
        b,
        h,
        s,
        d_head,
        B_r,
        B_c);

    // check CUDA error status (calls hipGetLastError())
    C10_CUDA_KERNEL_LAUNCH_CHECK();

    return std::make_tuple(out, l_hbm, m_hbm);
}